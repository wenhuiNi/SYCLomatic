
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const void *alpha,
          const void *a, hipDataType atype, int lda, long long int stridea,
          const void *b, hipDataType btype, int ldb, long long int strideb,
          const void *beta, void *c, hipDataType ctype, int ldc,
          long long int stridec, int group_count,
          hipblasComputeType_t computetype_computeType_t,
          hipDataType computetype_dataType, hipblasGemmAlgo_t algo) {
  // Start
  hipblasGemmStridedBatchedEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void **/, atype /*cudaDataType*/,
      lda /*int*/, stridea /*long long int*/, b /*const void **/,
      btype /*cudaDataType*/, ldb /*int*/, strideb /*long long int*/,
      beta /*const void **/, c /*void **/, ctype /*cudaDataType*/, ldc /*int*/,
      stridec /*long long int*/, group_count /*int*/,
      computetype_computeType_t /*cublasComputeType_t*/,
      algo /*cublasGemmAlgo_t*/);
  hipblasGemmStridedBatchedEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void **/, atype /*cudaDataType*/,
      lda /*int*/, stridea /*long long int*/, b /*const void **/,
      btype /*cudaDataType*/, ldb /*int*/, strideb /*long long int*/,
      beta /*const void **/, c /*void **/, ctype /*cudaDataType*/, ldc /*int*/,
      stridec /*long long int*/, group_count /*int*/,
      computetype_dataType /*cudaDataType*/, algo /*cublasGemmAlgo_t*/);
  // End
}
