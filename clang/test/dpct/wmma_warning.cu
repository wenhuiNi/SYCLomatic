// clang-format off
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0
// RUN: dpct --format-range=none -out-root %T/wmma_warning %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/wmma_warning/wmma_warning.dp.cpp --match-full-lines %s
#include <assert.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 16
#define N 16
#define K 16

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define M_TILES 16
#define N_TILES 16
#define K_TILES 16

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define WARP_SIZE 32
#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

__host__ void init_host_matrices(half *a, half *b, float *c) {
  for (int i = 0; i < M_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      a[i * K_GLOBAL + j] = (half)(rand() % 3);
    }
  }

  for (int i = 0; i < N_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      b[i * K_GLOBAL + j] = (half)(rand() % 3);
    }
  }

  for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
    c[t] = static_cast<float>(rand() % 3);
  }
}

__global__ void simple_wmma_gemm(half *a, half *b, float *c, float *d, int m_ld,
                                 int n_ld, int k_ld, float alpha, float beta) {
  // Leading dimensions. Packed with no transpositions.
  int lda = k_ld;
  int ldb = k_ld;
  int ldc = n_ld;

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  // Declare the fragments
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> type is not supported.
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::matrix_a type is not supported.
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::row_major type is not supported.
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::row_major>
      a_frag;
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> type is not supported.
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::matrix_b type is not supported.
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::col_major type is not supported.
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::col_major>
      b_frag;
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> type is not supported.
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::accumulator type is not supported.
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> type is not supported.
  // CHECK: DPCT1082:{{[0-9]+}}: Migration of nvcuda::wmma::accumulator type is not supported.
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
  // CHECK: DPCT1007:{{[0-9]+}}: Migration of nvcuda::wmma::fill_fragment is not supported.
  nvcuda::wmma::fill_fragment(acc_frag, 0.0f);

  // Loop over k
  for (int i = 0; i < k_ld; i += WMMA_K) {
    int aCol = i;
    int aRow = warpM * WMMA_M;
    int bCol = warpN * N;
    int bRow = i;

    // Bounds checking
    if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
      // Load the inputs
      // CHECK: DPCT1007:{{[0-9]+}}: Migration of nvcuda::wmma::load_matrix_sync is not supported.
      nvcuda::wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
      // CHECK: DPCT1007:{{[0-9]+}}: Migration of nvcuda::wmma::load_matrix_sync is not supported.
      nvcuda::wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

      // Perform the matrix multiplication
      // CHECK: DPCT1007:{{[0-9]+}}: Migration of nvcuda::wmma::mma_sync is not supported.
      nvcuda::wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cCol = warpN * WMMA_N;
  int cRow = warpM * WMMA_M;

  if (cRow < m_ld && cCol < n_ld) {
    // CHECK: DPCT1007:{{[0-9]+}}: Migration of nvcuda::wmma::load_matrix_sync is not supported.
    nvcuda::wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc,
                                   nvcuda::wmma::mem_row_major);

    // Store the output
    // CHECK: DPCT1007:{{[0-9]+}}: Migration of nvcuda::wmma::store_matrix_sync is not supported.
    nvcuda::wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc,
                                    nvcuda::wmma::mem_col_major);
  }
}

int main() {
  half *A_h = NULL;
  half *B_h = NULL;
  float *C_h = NULL;
  A_h = (half *)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
  B_h = (half *)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
  C_h = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  half *A = NULL;
  half *B = NULL;
  float *C = NULL;
  float *D = NULL;

  hipMalloc(reinterpret_cast<void **>(&A),
             sizeof(half) * M_GLOBAL * K_GLOBAL);
  hipMalloc(reinterpret_cast<void **>(&B),
             sizeof(half) * N_GLOBAL * K_GLOBAL);
  hipMalloc(reinterpret_cast<void **>(&C),
             sizeof(float) * M_GLOBAL * N_GLOBAL);
  hipMalloc(reinterpret_cast<void **>(&D),
             sizeof(float) * M_GLOBAL * N_GLOBAL);

  assert(((unsigned long long)A) % 128 == 0);
  assert(((unsigned long long)B) % 128 == 0);
  assert(((unsigned long long)C) % 128 == 0);
  assert(((unsigned long long)D) % 128 == 0);

  init_host_matrices(A_h, B_h, C_h);

  printf("Preparing data for GPU...\n");

  hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL,
             hipMemcpyHostToDevice);
  hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL,
             hipMemcpyHostToDevice);
  hipMemcpy(C, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL,
             hipMemcpyHostToDevice);
  hipMemset(D, 0, sizeof(float) * M_GLOBAL * N_GLOBAL);

  const float alpha = 1.1f;
  const float beta = 1.2f;

  dim3 gridDim;
  dim3 blockDim;

  // blockDim.x must be a multple of warpSize
  // 128x4 means we have 16 warps and a block computes a 64x64 output tile
  blockDim.x = 128;
  blockDim.y = 4;

  gridDim.x = (M_GLOBAL + (WMMA_M * blockDim.x / 32 - 1)) /
              (WMMA_M * blockDim.x / 32);
  gridDim.y = (N_GLOBAL + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  printf("Computing... using simple_wmma_gemm kernel\n");
  simple_wmma_gemm<<<gridDim, blockDim>>>(A, B, C, D, M_GLOBAL, N_GLOBAL,
                                          K_GLOBAL, alpha, beta);
  hipDeviceSynchronize();

  free(A_h);
  free(B_h);
  free(C_h);
  hipFree(reinterpret_cast<void *>(A));
  hipFree(reinterpret_cast<void *>(B));
  hipFree(reinterpret_cast<void *>(C));
  hipFree(reinterpret_cast<void *>(D));

  return 0;
}
// clang-format on
